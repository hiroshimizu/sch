#include "hip/hip_runtime.h"
#pragma once  // ヘッダファイルにはこれを書く

#ifndef __SCHRO_CONSTANT__
#define __SCHRO_CONSTANT__
#include<string>
#include<iostream>
#include<fstream>
#include<sstream>
#include<limits>
#include<map>
#include<cstdio>

const REAL pi = 3.1415926535897932384626433832795028841971693993751;


// 文字列の空白を取り除く
std::string trim(std::string str)
{	std::stringstream ss(str);
	std::string s;
	ss >> s;
	return s;
}

#define READ_VALUE(NAME,LIST) \
	NAME=read_value(NAME,#NAME,LIST)
template<class T>
T read_value(
	T& var, std::string name,
	std::map<std::string,std::string>& param_list)
{	typedef std::map<std::string,std::string> list_type;

	// 読み込んだパラメータリストにあるか探す
	const list_type::iterator itr=param_list.find(name);
	if(itr!=param_list.end()) {
		// 適切な型に変換して代入
		std::string str = itr->second;
		std::stringstream ss(str);
		ss >> var;
	}
	else {
		// リストになければNaNにしておく(整数の場合は0)
		var = std::numeric_limits<T>::quiet_NaN();
		std::cout << "Error: Parameter is not found in file" << std::endl;
	}
	return var;
}

std::map<std::string,std::string>
read_parameter_list(std::string filename)
{	using std::string;
	std::ifstream ifs(filename.c_str());
	std::map<string,string> parameter_list;
	while(ifs) {
		// 一行読み込み
		string line;
		std::getline(ifs, line);

		// # 以下はコメント扱いとして消去
		string::size_type pos=line.find("#");
		if(pos!=string::npos) {
			line.erase(line.begin()+pos,line.end());
		}

		// "="で分割
		std::stringstream ss(line);
		string first, second;
		std::getline(ss, first,'=');
		std::getline(ss,second,'=');

		// 単語前後のスペースを除去
		first = trim(first);
		second= trim(second);

		// 取得結果を保存
		if(!first.empty() && !second.empty()) {
			parameter_list[first] = second;
		}
	}
	return parameter_list;
}

bool read_parameter(std::string filename,
		int& N_rec, int& N_rec_rho, int& N_step, REAL& hx, REAL& hy, REAL& Lx, REAL& Ly,
		REAL& omg, REAL& v0x, REAL& v0y, REAL& Bz, REAL& m, REAL& q, REAL& x0, REAL& y0,
		REAL& vSIZE, REAL& v0_, REAL& Bz_, REAL& mp_, REAL& e_, REAL& h_bar_)
{	// 定数読み込み
	std::map<std::string,std::string>
		parameter_list = read_parameter_list(filename);
	if(parameter_list.empty())return false;

	// (READ_VALUEは宣言していない変数を使うとエラーとなる)
	READ_VALUE(N_rec	 ,parameter_list);
	READ_VALUE(N_rec_rho ,parameter_list);
	READ_VALUE(N_step	 ,parameter_list);
	READ_VALUE(hx      	 ,parameter_list);
	READ_VALUE(hy      	 ,parameter_list);
	READ_VALUE(Lx      	 ,parameter_list);
	READ_VALUE(Ly      	 ,parameter_list);
	READ_VALUE(omg    	 ,parameter_list);
	READ_VALUE(v0x    	 ,parameter_list);
	READ_VALUE(v0y    	 ,parameter_list);
	READ_VALUE(Bz		 ,parameter_list);
	READ_VALUE(m		 ,parameter_list);
	READ_VALUE(q		 ,parameter_list);
	READ_VALUE(x0   	 ,parameter_list);
	READ_VALUE(y0   	 ,parameter_list);
	READ_VALUE(vSIZE  	 ,parameter_list);
	READ_VALUE(v0_  	 ,parameter_list);
	READ_VALUE(Bz_    	 ,parameter_list);
	READ_VALUE(mp_  	 ,parameter_list);
	READ_VALUE(e_     	 ,parameter_list);
	READ_VALUE(h_bar_ 	 ,parameter_list);

	return true;
}
#endif
