#include "hip/hip_runtime.h"
#pragma once  // ヘッダファイルにはこれを書く

	__constant__ int   Nb_d, Nx2_d, Ny2_d, Nb3__d, Nx_d, Ny_d;
	__constant__ REAL  sgm2_d, pi_d, hx_d, hy_d, hh_d, omg_d, x0_d, y0_d, Bz_d, q_d;
	__constant__ CMPLX beta_d, gamma_d, zeta_d, xi_d;

void SetConst
	(	int Nb, int Nx2, int Ny2, int Nb3_, int Nx, int Ny, REAL sgm2, REAL pi, REAL hx, REAL hy,
		REAL hh, REAL omg, REAL x0, REAL y0, REAL Bz, REAL q, CMPLX beta, CMPLX gamma, CMPLX zeta, CMPLX xi)
{	(hipMemcpyToSymbol(HIP_SYMBOL(    Nb_d),    &Nb, sizeof(Nb)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(   Nx2_d),   &Nx2, sizeof(Nx2)  ));
	(hipMemcpyToSymbol(HIP_SYMBOL(   Ny2_d),   &Ny2, sizeof(Ny2)  ));
	(hipMemcpyToSymbol(HIP_SYMBOL(  Nb3__d),  &Nb3_, sizeof(Nb3_) ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    Nx_d),    &Nx, sizeof(Nx)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    Ny_d),    &Ny, sizeof(Ny)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(  sgm2_d),  &sgm2, sizeof(sgm2) ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    pi_d),    &pi, sizeof(pi)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    hx_d),    &hx, sizeof(hx)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    hy_d),    &hy, sizeof(hy)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    hh_d),    &hh, sizeof(hh)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(   omg_d),   &omg, sizeof(omg)  ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    x0_d),    &x0, sizeof(x0)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    y0_d),    &y0, sizeof(y0)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    Bz_d),    &Bz, sizeof(Bz)   ));
	(hipMemcpyToSymbol(HIP_SYMBOL(     q_d),     &q, sizeof(q)    ));
	(hipMemcpyToSymbol(HIP_SYMBOL(  beta_d),  &beta, sizeof(beta) ));
	(hipMemcpyToSymbol(HIP_SYMBOL( gamma_d), &gamma, sizeof(gamma)));
	(hipMemcpyToSymbol(HIP_SYMBOL(  zeta_d),  &zeta, sizeof(zeta) ));
	(hipMemcpyToSymbol(HIP_SYMBOL(    xi_d),    &xi, sizeof(xi)   ));
}

#ifdef FieldParticle
__global__ void PHI ( CMPLX *psi, CMPLX *phi, REAL *u, int N_field )
#else
__global__ void PHI ( CMPLX *psi, CMPLX *phi, int N_field )
#endif
{	CMPLX dpsi_dx, dpsi_dy, d2psi_dx2, d2psi_dy2;
	CMPLX BETA_ij, GAMMA_ij, ZETA_ij, XI_ij;
	REAL  qAx, qAy;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int ix = idx % Nx2_d;
	int iy = idx / Nx2_d;
	int II = ix  + Nx2_d * iy;
	//phi[II] = 0;
	if ( 0 < ix && ix < Nx2_d - 1 && 0 < iy && iy <= Nb_d )
	{	REAL x  = ( ix - Nx_d ) * hx_d;
		REAL y  = ( iy - Ny_d + N_field * Nb_d ) * hy_d;
		qA ( x, y, Bz_d, q_d, &qAx, &qAy );
		FDM ( II, psi, hx_d, hy_d, Nx2_d, &dpsi_dx, &dpsi_dy, &d2psi_dx2, &d2psi_dy2 );

		#ifdef CUDACOMPLEX_H
			BETA_ij   =  beta_d * ( d2psi_dx2 * d2psi_dy2 );
			GAMMA_ij  = gamma_d * ( qAx * dpsi_dx + qAy * dpsi_dy );
			ZETA_ij   =  zeta_d * ( Square( qAx ) + Square( qAy ) ) * psi[II];
			#ifdef FieldParticle
				XI_ij = xi_d * u[ II + N_field * Nb_d ] * psi[II];
			#else
				XI_ij = xi_d *        potential( x, y, q_d ) * psi[II];
			#endif
			phi[II] = psi[II] + BETA_ij + GAMMA_ij - ZETA_ij - XI_ij;
		#else
			BETA_ij   = hipCmul(  beta_d, hipCadd( d2psi_dx2, d2psi_dy2 ) );
			GAMMA_ij  = hipCmul( gamma_d, hipCadd( hipCmul( mkCMPLX( qAx, 0 ), dpsi_dx ), hipCmul( mkCMPLX( qAy, 0 ), dpsi_dy ) ) );
			ZETA_ij   = hipCmul(  zeta_d, hipCmul( mkCMPLX( Square( qAx ) + Square( qAy ), 0 ), psi[II] ) );
			#ifdef FieldParticle
				XI_ij = hipCmul( xi_d, hipCmul( mkCMPLX( u[ II + N_field * Nb_d ], 0 ), psi[II] ) );
			#else
				XI_ij = hipCmul( xi_d, hipCmul( mkCMPLX( potential( x, y, q_d ), 0 ), psi[II] ) );
			#endif
			phi[II] = hipCsub( hipCsub( hipCadd( hipCadd( psi[II], BETA_ij ), GAMMA_ij ), ZETA_ij ), XI_ij );
		#endif
	}
	__syncthreads();
}


#ifdef FieldParticle
__global__ void SOR ( CMPLX *psi, CMPLX *phi, REAL *res, REAL *u, int N_field )
#else
__global__ void SOR ( CMPLX *psi, CMPLX *phi, REAL *res, int N_field )
#endif
{	CMPLX dpsi_dx, dpsi_dy, d2psi_dx2, d2psi_dy2;
	CMPLX ALPHA_ij, BETA_ij, GAMMA_ij, ZETA_ij, XI_ij;
	CMPLX ResC, tmpC = {0,0};
	REAL  qAx, qAy;
	REAL  tmpR = 0;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int ix = idx % Nx2_d;
	int iy = idx / Nx2_d;
	int II = ix + Nx2_d * iy;
	if( 0 < ix && ix < Nx2_d - 1 && 0 < iy && iy <= Nb_d)
	{	REAL x = ( ix - Nx_d ) * hx_d;
		REAL y = ( iy - Ny_d + N_field * Nb_d ) * hy_d;
		qA ( x, y, Bz_d, q_d, &qAx, &qAy );
		FDM ( II, psi, hx_d, hy_d, Nx2_d, &dpsi_dx, &dpsi_dy, &d2psi_dx2, &d2psi_dy2 );

		#ifdef CUDACOMPLEX_H
			BETA_ij  =  beta_d * ( d2psi_dx2 + d2psi_dy2 );
			GAMMA_ij = gamma_d * ( qAx * dpsi_dx + qAy * dpsi_dy );
			ZETA_ij  =  zeta_d * ( Square( qAx ) + Square( qAy ) ) * psi[II];
			#ifdef FieldParticle
				XI_ij = xi_d * u[ II + N_field * Nb_d ] * psi[II];
				ALPHA_ij = 1	+ beta_d * 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) )
					         	+ zeta_d *     (     Square( qAx  ) +     Square( qAy  ) )
					         	+   xi_d * u[ II + N_field * Nb_d ];
			#else
				XI_ij = hipCmul( xi_d, hipCmul( mkCMPLX( potential( x, y, q_d ), 0 ), psi[II] ) );
				ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
					hipCmul( beta_d, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
					hipCmul( zeta_d, mkCMPLX(           Square( qAx )  +     Square( qAy  )  , 0 ) ) ),
					hipCmul(   xi_d, mkCMPLX( potential( x, y, q_d )                              , 0 ) ) );
			#endif
			ResC = phi[II] - psi[II] + BETA_ij + GAMMA_ij - ZETA_ij - XI_ij;
			tmpC = omg_d / ALPHA_ij * ResC;
			tmpR = Square( ResC.real() ) + Square( ResC.imag() );
		#else
			BETA_ij  = hipCmul(  beta_d, hipCadd( d2psi_dx2, d2psi_dy2 ) );
			GAMMA_ij = hipCmul( gamma_d, hipCadd( hipCmul( mkCMPLX( qAx, 0 ), dpsi_dx ), hipCmul( mkCMPLX( qAy, 0 ), dpsi_dy ) ) );
			ZETA_ij  = hipCmul(  zeta_d, hipCmul( mkCMPLX( Square( qAx ) + Square( qAy ), 0 ), psi[II] ) );
			#ifdef FieldParticle
				XI_ij = hipCmul( xi_d, hipCmul( mkCMPLX( u[ II + N_field * Nb_d ], 0 ), psi[II] ) );
				ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
					hipCmul( beta_d, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
					hipCmul( zeta_d, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
					hipCmul( xi_d, mkCMPLX( u[ II + N_field * Nb_d ], 0 ) ) );
			#else
				XI_ij = hipCmul( xi_d, hipCmul( mkCMPLX( potential( x, y, q_d ), 0 ), psi[II] ) );
				ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
					hipCmul( beta_d, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
					hipCmul( zeta_d, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
					hipCmul( xi_d, mkCMPLX( potential( x, y, q_d ), 0 ) ) );
			#endif
			ResC = hipCsub( phi[II], hipCsub( psi[II], hipCadd( BETA_ij, hipCsub( hipCsub( GAMMA_ij, ZETA_ij ), XI_ij ) ) ) ) ;
			tmpC = hipCmul( hipCdiv( mkCMPLX( omg_d, 0 ), ALPHA_ij ), ResC );
			tmpR = Square( hipCabs( ResC ) );
		#endif
	__syncthreads();
	#ifdef CUDACOMPLEX_H
		psi[II] =         psi[II] + tmpC  ;
	#else
		psi[II] = hipCadd( psi[II],  tmpC );
	#endif
	res[II] = tmpR ;
	}
}


#ifdef FieldParticle
__global__ void sor0 ( CMPLX *psi, CMPLX *phi, REAL *res, REAL *u, int N_field )
#else
__global__ void sor0 ( CMPLX *psi, CMPLX *phi, REAL *res, int N_field )
#endif
{	int iy = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ CMPLX beta_s, gamma_s, zeta_s, xi_s;
	if ( threadIdx.x == 0 )
	{	beta_s  =  beta_d;
		gamma_s = gamma_d;
		zeta_s  =  zeta_d;
		xi_s    =    xi_d;
	}	__syncthreads();

	CMPLX dpsi_dx, dpsi_dy, d2psi_dx2, d2psi_dy2;
	CMPLX ALPHA_ij, BETA_ij, GAMMA_ij, ZETA_ij, XI_ij;
	CMPLX ResC;
	REAL  qAx, qAy;
	if ( iy % 2 == 0 )
	{	if ( 0 < iy && iy <= Nb_d )
		{	REAL y = ( iy - Ny_d + N_field * Nb_d ) * hy_d;
			for( int ix = 1; ix < Nx2_d - 1; ix++ )
			{	int II = ix + Nx2_d * iy;
				REAL x = ( ix - Nx_d ) * hx_d;
				qA ( x, y, Bz_d, q_d, &qAx, &qAy );
				FDM ( II, psi, hx_d, hy_d, Nx2_d, &dpsi_dx, &dpsi_dy, &d2psi_dx2, &d2psi_dy2 );

			#ifdef CUDACOMPLEX_H
				 BETA_ij =  beta_s * ( d2psi_dx2 + d2psi_dy2 );
				GAMMA_ij = gamma_s * ( qAx * dpsi_dx + qAy * dpsi_dy );
				 ZETA_ij =  zeta_s * ( Square( qAx ) + Square( qAy ) ) * psi[II];
				#ifdef FieldParticle
					XI_ij = xi_s * u[ II + N_field * Nb_d ] * psi[II];
					ALPHA_ij = 1	+ beta_s * 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) )
						         	+ zeta_s *     (     Square( qAx  ) +     Square( qAy  ) )
						          	+   xi_s * u[ II + N_field * Nb_d ];
				#else
					XI_ij = xi_s * potential( x, y, q_d ) * psi[II];
					ALPHA_ij = 1 	+ beta_s * 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) )
						          	+ zeta_s *     (     Square( qAx  ) +     Square( qAy  ) )
						          	+   xi_s * potential( x, y, q_d );
				#endif
				ResC = phi[II] - psi[II] + BETA_ij + GAMMA_ij - ZETA_ij - XI_ij ;
				psi[II] = psi[II] + omg_d / ALPHA_ij * ResC;
				res[II] = Square( ResC.real() );
			#else
				BETA_ij  = hipCmul(  beta_s, hipCadd( d2psi_dx2, d2psi_dy2 ) );
				GAMMA_ij = hipCmul( gamma_s, hipCadd( hipCmul( mkCMPLX( qAx, 0 ), dpsi_dx ), hipCmul( mkCMPLX( qAy, 0 ), dpsi_dy ) ) );
				ZETA_ij  = hipCmul(  zeta_s, hipCmul( mkCMPLX( Square( qAx ) + Square( qAy ), 0 ), psi[II] ) );
				#ifdef FieldParticle
					XI_ij = hipCmul( xi_s, hipCmul( mkCMPLX( u[ II + N_field * Nb_d ], 0 ), psi[II] ) );
					ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
						hipCmul( beta_s, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
						hipCmul( zeta_s, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
						hipCmul( xi_s, mkCMPLX( u[ II + N_field * Nb_d ], 0 ) ) );
				#else
					XI_ij = hipCmul( xi_s, hipCmul( mkCMPLX( potential( x, y, q_d ), 0 ), psi[II] ) );
					ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
						hipCmul( beta_s, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
						hipCmul( zeta_s, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
						hipCmul( xi_s, mkCMPLX( potential( x, y, q_d ), 0 ) ) );
				#endif
				ResC = hipCsub( phi[II], hipCsub( psi[II], hipCadd( BETA_ij, hipCsub( hipCsub( GAMMA_ij, ZETA_ij ), XI_ij ) ) ) ) ;
				psi[II] = hipCadd( psi[II], hipCmul( hipCdiv( mkCMPLX( omg_d, 0 ), ALPHA_ij ), ResC ) );
				res[II] = Square( hipCabs( ResC ) );
			#endif
			}
			res[            Nx2_d * iy] = 0;	//	ix = 0
			res[Nx2_d - 1 + Nx2_d * iy] = 0;	//	ix = Nx2_d - 1
		}
		else if ( iy == 0 || iy == Nb_d + 1 )
		{	for( int ix = 0; ix < Nx2_d; ix++ )
			{	int II  = ix + Nx2_d * iy;
				res[II] = 0;
			}
		}
	}
	__syncthreads();
}

#ifdef FieldParticle
__global__ void sor1 ( CMPLX *psi, CMPLX *phi, REAL *res, REAL *u, int N_field )
#else
__global__ void sor1 ( CMPLX *psi, CMPLX *phi, REAL *res, int N_field )
#endif
{	int iy = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ CMPLX beta_s, gamma_s, zeta_s, xi_s;
	if ( threadIdx.x == 0 )
	{	beta_s  =  beta_d;
		gamma_s = gamma_d;
		zeta_s  =  zeta_d;
		xi_s    =    xi_d;
	}	__syncthreads();

	CMPLX dpsi_dx, dpsi_dy, d2psi_dx2, d2psi_dy2;
	CMPLX ALPHA_ij, BETA_ij, GAMMA_ij, ZETA_ij, XI_ij;
	CMPLX ResC;
	REAL  qAx, qAy;

	if ( iy % 2 == 1 )
	{	if ( 0 < iy && iy <= Nb_d )
		{	REAL y = ( iy - Ny_d + N_field * Nb_d ) * hy_d;
			for( int ix = 1; ix < Nx2_d - 1; ix++ )
			{	int II = ix + Nx2_d * iy;
				REAL x = ( ix - Nx_d ) * hx_d;
				qA ( x, y, Bz_d, q_d, &qAx, &qAy );
				FDM ( II, psi, hx_d, hy_d, Nx2_d, &dpsi_dx, &dpsi_dy, &d2psi_dx2, &d2psi_dy2 );

			#ifdef CUDACOMPLEX_H
				BETA_ij  =  beta_s * ( d2psi_dx2 + d2psi_dy2 );
				GAMMA_ij = gamma_s * ( qAx * dpsi_dx + qAy * dpsi_dy );
				ZETA_ij  =  zeta_s * ( Square( qAx ) + Square( qAy ) ) * psi[II];
				#ifdef FieldParticle
					XI_ij = xi_s * u[ II + N_field * Nb_d ] * psi[II];
					ALPHA_ij = 1	+ beta_s * 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) )
						          	+ zeta_s *     (     Square( qAx  ) +     Square( qAy  ) )
						          	+   xi_s * u[ II + N_field * Nb_d ];
				#else
					XI_ij = xi_s * potential( x, y, q_d ) * psi[II];
					ALPHA_ij = 1 	+ beta_s * 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) )
						          	+ zeta_s *     (     Square( qAx  ) +     Square( qAy  ) )
						          	+   xi_s * potential( x, y, q_d );
				#endif
				ResC = phi[II] - psi[II] + BETA_ij + GAMMA_ij - ZETA_ij - XI_ij;
				psi[II] = psi[II] + omg_d / ALPHA_ij * ResC;
				res[II] = Square( ResC.real() ) + Square( ResC.imag() );
			#else
				BETA_ij  = hipCmul(  beta_s, hipCadd( d2psi_dx2, d2psi_dy2 ) );
				GAMMA_ij = hipCmul( gamma_s, hipCadd( hipCmul( mkCMPLX( qAx, 0 ), dpsi_dx ), hipCmul( mkCMPLX( qAy, 0 ), dpsi_dy ) ) );
				ZETA_ij  = hipCmul(  zeta_s, hipCmul( mkCMPLX( Square( qAx ) + Square( qAy ), 0 ), psi[II] ) );
				#ifdef FieldParticle
					XI_ij = hipCmul( xi_s, hipCmul( mkCMPLX( u[ II + N_field * Nb_d ], 0 ), psi[II] ) );
					ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
						hipCmul( beta_s, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
						hipCmul( zeta_s, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
						hipCmul( xi_s, mkCMPLX( u[ II + N_field * Nb_d ], 0 ) ) );
				#else
					XI_ij = hipCmul( xi_s, hipCmul( mkCMPLX( potential( x, y, q_d ), 0 ), psi[II] ) );
					ALPHA_ij = hipCadd( hipCadd( hipCadd( mkCMPLX( 1, 0 ),
						hipCmul( beta_s, mkCMPLX( 2 * ( 1 / Square( hx_d ) + 1 / Square( hy_d ) ), 0 ) ) ),
						hipCmul( zeta_s, mkCMPLX( Square( qAx ) + Square( qAy ), 0 ) ) ),
						hipCmul( xi_s, mkCMPLX( potential( x, y, q_d ), 0 ) ) );
				#endif
				ResC = hipCsub( phi[II], hipCsub( psi[II], hipCadd( BETA_ij, hipCsub( hipCsub( GAMMA_ij, ZETA_ij ), XI_ij ) ) ) ) ;
				psi[II] = hipCadd( psi[II], hipCmul( hipCdiv( mkCMPLX( omg_d, 0 ), ALPHA_ij ), ResC ) );
				res[II] = Square( hipCabs( ResC ) );
			#endif
			}
			res[            Nx2_d * iy] = 0;	//	ix = 0
			res[Nx2_d - 1 + Nx2_d * iy] = 0;	//	ix = Nx2_d - 1
		}
		else if ( iy == 0 || iy == Nb_d + 1 )
		{	for( int ix = 0; ix < Nx2_d; ix++ )
			{	int II  = ix + Nx2_d * iy;
				res[II] = 0;
			}
		}
	}
	__syncthreads();
}
