#pragma once  // ヘッダファイルにはこれを書く

void EXPECT ( CMPLX *psi, REAL *x, REAL *y,
#ifdef FieldParticle
	REAL *u,
#endif
	CMPLX *mvx_av,  CMPLX *mvy_av,  CMPLX *Px,    REAL  *E,      CMPLX *var_mv,
	CMPLX *var_mvx, CMPLX *var_mvy, CMPLX *var_P, CMPLX *var_Px, CMPLX *var_Py,
	REAL *x_av,     REAL *y_av,     REAL *var_r,  REAL *var_x,   REAL *var_y, 
	REAL *err,      REAL *qAx_av,
	REAL Bz, REAL m, REAL q, int Nx2, int Ny2, REAL hx, REAL hy, REAL h_bar )
{	REAL  x_ij, y_ij, rho_ij, qAx, qAy;
	CMPLX dpsi_dx, dpsi_dy, dpsi2dx, dpsi2dy;
	CMPLX psi_cj;
	CMPLX Px_av = {0,0}, Py_av = {0,0}, Px2av = {0,0}, Py2av = {0,0};
	CMPLX mv2av;
	CMPLX mvx2av;
	CMPLX mvy2av;
	CMPLX qAxPx_av = {0,0}; CMPLX qAyPy_av = {0,0};
	REAL  qAy_av = 0, qAx2av = 0, qAy2av = 0;
	REAL  x2av = 0, y2av = 0;
	CMPLX K_av, E_av;
	REAL  U_av = 0;
	REAL  hh = hx * hy;

	/* Calculation */
	/*	Px_av, Px2av, Py_av, Py2av, 
		qAx_av,qAx2av,qAy_av,qAy2av, 
		qAxPx_av, qAyPy_av, U_av	*/
	*qAx_av = 0;
	for(		int iy = 1; iy < Ny2 - 1; iy++) { y_ij = y[iy];
		for(	int ix = 1; ix < Nx2 - 1; ix++) { x_ij = x[ix];
			int II = ix + Nx2 * iy;
			FDM ( II, psi, hx, hy, Nx2, &dpsi_dx, &dpsi_dy, &dpsi2dx, &dpsi2dy );
			#ifdef CUDACOMPLEX_H
				psi_cj = ~psi[II];
				rho_ij = ( psi_cj * psi[II] ).real();

				Px_av = Px_av + psi_cj * dpsi_dx;
				Py_av = Py_av + psi_cj * dpsi_dy;
				Px2av = Px2av + psi_cj * dpsi2dx;
				Py2av = Py2av + psi_cj * dpsi2dy;
				qA ( x_ij, y_ij, Bz, q, &qAx, &qAy );	// qAy == 0が来ているはず
				*qAx_av +=         qAx   * rho_ij;
				 qAy_av +=         qAy   * rho_ij;	// == 0のはず
				qAx2av  += Square( qAx ) * rho_ij;
				qAy2av  += Square( qAy ) * rho_ij;	// == 0のはず
				qAxPx_av = qAxPx_av + psi_cj * ( qAx * dpsi_dx );
				qAyPy_av = qAyPy_av + psi_cj * ( qAy * dpsi_dy );	// == 0のはず
			#else
				rho_ij = Square( hipCabs( psi[II] ) );
				psi_cj = hipConj(         psi[II]   );
				Px_av  = hipCadd( Px_av, hipCmul( psi_cj, dpsi_dx ) );
				Py_av  = hipCadd( Py_av, hipCmul( psi_cj, dpsi_dy ) );
				Px2av  = hipCadd( Px2av, hipCmul( psi_cj, dpsi2dx ) );
				Py2av  = hipCadd( Py2av, hipCmul( psi_cj, dpsi2dy ) );
				qA ( x_ij, y_ij, Bz, q, &qAx, &qAy );	// qAy == 0が来ているはず
				*qAx_av +=         qAx   * rho_ij;
				 qAy_av +=         qAy   * rho_ij;	// == 0のはず
				qAx2av +=  Square( qAx ) * rho_ij;
				qAy2av +=  Square( qAy ) * rho_ij;	// == 0のはず
				qAxPx_av = hipCadd( qAxPx_av, hipCmul( psi_cj, hipCmul( mkCMPLX(qAx,0), dpsi_dx )));
				qAyPy_av = hipCadd( qAyPy_av, hipCmul( psi_cj, hipCmul( mkCMPLX(qAy,0), dpsi_dy )));	// == 0のはず
			#endif
			#ifndef FieldParticle
				U_av += potential( x_ij, y_ij, q ) * rho_ij;
			#else
				U_av +=                   u[II] * rho_ij;
			#endif
		}
	}

	/* Calculation */
	/*	x_av, x2av, y_av, y2av, err */
	*x_av = 0; *y_av = 0; *err = 0;
	for(		int iy = 0; iy < Ny2; iy++) { y_ij = y[iy];
		for(	int ix = 0; ix < Nx2; ix++) { x_ij = x[ix];
			int II = ix + Nx2 * iy;
			#ifdef CUDACOMPLEX_H
				rho_ij = psi[II].abs2();	//rho_ij = ( ~psi[II] * psi[II] ).real();
			#else
				rho_ij = Square( hipCabs( psi[II] ) );
			#endif
			*x_av += 		  x_ij   * rho_ij;
			*y_av += 		  y_ij   * rho_ij;
			x2av  += Square( x_ij ) * rho_ij;
			y2av  += Square( y_ij ) * rho_ij;
			*err	+= 				     rho_ij;
		}
	}
	
	#ifdef CUDACOMPLEX_H
		CMPLX I = {0, 1};
	//	CMPLX I = make_doublecomplex(0, 1);
	//	CMPLX I = make_REALcomplex(0, 1);
		Px_av = - h_bar * hh * I * Px_av;	// 運動の恒量for H = H( y, \vec v)
		Py_av = - h_bar * hh * I * Py_av;
		Px2av = - h_bar * h_bar * hh * Px2av;
		Py2av = - h_bar * h_bar * hh * Py2av;
		qAxPx_av = hh * qAxPx_av;
		qAyPy_av = hh * qAyPy_av;
	#else
		Px_av  = hipCmul( mkCMPLX( 0, - h_bar * hh ), Px_av );	// 運動の恒量for H = H( y, \vec v)
		Py_av  = hipCmul( mkCMPLX( 0, - h_bar * hh ), Py_av );
		Px2av  = hipCmul( mkCMPLX( - h_bar * h_bar * hh, 0 ), Px2av );
		Py2av  = hipCmul( mkCMPLX( - h_bar * h_bar * hh, 0 ), Py2av );
		qAxPx_av = hipCmul( mkCMPLX( hh, 0 ), qAxPx_av );
		qAyPy_av = hipCmul( mkCMPLX( hh, 0 ), qAyPy_av );
	#endif
	*qAx_av *= hh;
	 qAy_av *= hh;
	 qAx2av *= hh;
	 qAy2av *= hh;

	U_av  *= hh;

	*x_av *= hh;
	*y_av *= hh;
	 x2av *= hh;
	 y2av *= hh;
	*err  *= hh;
	
	/* Calculation */
	/*	mvx_av, mvx2av, mvy_av, mvy2_av, mv_av,
		K_av, E_av, and variances */
	#ifdef CUDACOMPLEX_H
		*mvx_av = Px_av - *qAx_av;
		*mvy_av = Py_av -  qAy_av;
		 mvx2av = Px2av + qAx2av + 2 * h_bar * I * qAxPx_av;	//	Axがxの関数でないため、単純に2乗を展開した形
		 mvy2av = Py2av + qAy2av + 2 * h_bar * I * qAyPy_av;	//	同上
		 mv2av  = mvx2av + mvy2av;
		*var_mvx= mvx2av       	- ( *mvx_av * *mvx_av );
		*var_mvy= mvy2av       	- ( *mvy_av * *mvy_av ); 
		*var_mv = var_mvx + var_mvy;
		*var_Px = Px2av - ( Px_av	* Px_av );
		*var_Py = Py2av - ( Py_av	* Py_av );
		*var_P  = *var_Px + *var_Py;
		K_av	  = mv2av / ( 2 * m );
		E_av    = K_av + U_av;
	#else
		*mvx_av = hipCsub( Px_av, mkCMPLX( *qAx_av, 0 ) );
		*mvy_av = hipCsub( Py_av, mkCMPLX(  qAy_av, 0 ) );
		 mvx2av = hipCadd( Px2av, hipCadd( mkCMPLX( qAx2av, 0 ), hipCmul( mkCMPLX( 0, 2 * h_bar ), qAxPx_av ) ) );
		 mvy2av = hipCadd( Py2av, hipCadd( mkCMPLX( qAy2av, 0 ), hipCmul( mkCMPLX( 0, 2 * h_bar ), qAyPy_av ) ) );
		 mv2av  = hipCadd( mvx2av, mvy2av);
		*var_mvx= hipCsub( mvx2av, hipCmul( *mvx_av, *mvx_av ) );
		*var_mvy= hipCsub( mvy2av, hipCmul( *mvy_av, *mvy_av ) );
		*var_mv = hipCadd( *var_mvx, *var_mvy );
		*var_Px = hipCsub( Px2av, hipCmul( Px_av, Px_av ) );
		*var_Py = hipCsub( Py2av, hipCmul( Py_av, Py_av ) );
		*var_P  = hipCadd( *var_Px, *var_Py );
		K_av	= hipCdiv( mv2av, mkCMPLX( 2 * m, 0 ) );
		E_av	= hipCadd( K_av, mkCMPLX( U_av, 0 ) );
	#endif
	*var_x  = x2av - Square( *x_av );
	*var_y  = y2av - Square( *y_av );
	*var_r  = *var_x + *var_y;
	*err   -= 1;
	
	*Px	= Px_av;
	#ifdef CUDACOMPLEX_H
		*E		= E_av.real();
	#else
		*E		= hipCreal( E_av );
	#endif
	/* FOR Check */
//	printf("U_av, K_av, E_av: %25.16e, %25.16e, %25.16e\n", U_av,hipCreal( K_av), hipCreal(E_av));
	/* END Check */
}
