#include "hip/hip_runtime.h"
#pragma once  // ヘッダファイルにはこれを書く

__device__ __host__
static inline REAL X ( REAL x ) { return 0; }

__device__ __host__
static inline REAL Y ( REAL y ) {
	//const REAL LB_INV = 1e-6;	// 磁場の勾配長の逆数、0のとき一様磁場
	//const REAL LB_INV = 1e-5;	// 磁場の勾配長の逆数、0のとき一様磁場
	const REAL LB_INV = 0;	// 磁場の勾配長の逆数、0のとき一様磁場
	return y * ( 1 - LB_INV / 2 * y);
}

__device__ __host__ void qA ( REAL x, REAL y, REAL Bz, REAL q, REAL* qAx, REAL* qAy )
{	*qAx = - q * Bz * Y(y);
	*qAy =   q * Bz * X(x);
}


#ifdef FieldParticle
extern "C"
{	void  hiab_();
	void  dehint_(REAL *, REAL *, REAL *, REAL *);
}

void PotEne
	(	REAL *u, const REAL *x, const REAL *y,
		const REAL q, const REAL sgm, const REAL Eps0, const int Nx2, const int Ny2
	)
{	const int  nf = 1;	// number of field particles
	REAL xf[nf], yf[nf], qf[nf];
	REAL zero = 0;		// lower bound of integration (積分区間は[a,b]=[0, infinity])
	REAL EPS  = 1e-12;	// absolute error tolerance
	REAL V;

	xf[0] = 0; yf[0] = 0; qf[0] = 3e-5;
	hiab_();
	for(		int iy = 0; iy < Ny2; iy++) { REAL y_ij = y[iy];
		for(	int ix = 0; ix < Nx2; ix++) { REAL x_ij = x[ix];
			int II = ix + Nx2 * iy;
			REAL phi_ij = 0;
			for( int f = 0; f < nf; f++)
			{	REAL R = sqrt( Square( x_ij - xf[f] ) + Square( y_ij - yf[f] ) );
				REAL eta = R / sgm;
				dehint_ ( &eta, &zero, &EPS, &V );
				phi_ij += qf[f] * V;
			}
			phi_ij =     phi_ij / ( Square( pi ) * Eps0 * sgm );
			u[II]  = q * phi_ij;
		}
	}
}
#else
	__device__ __host__ REAL potential ( REAL x, REAL y, REAL q )
	{
	//	return 0;
		#ifndef _UFM_
		const REAL LE4_INV = INP_NUM_;
		#else
		const REAL LE4_INV = 0;
		#endif
		const REAL Ey = 1;
		REAL y2 = y * y;
		return - q * Ey * y2 * y2 * y / 5 * LE4_INV;
	//	REAL Ey = 0;
	//	REAL Ey = 1;
	//	REAL Ey = 1e-4;
	//	REAL Ey = 1e-5;
	//	REAL Ey = 1e-6;
		//REAL LE_INV = 0;

	//	REAL k_E = 0.1;
	//	REAL k_E = 0.5;
	//	REAL k_E = 1;
	//	REAL k_E = 2;
	//	REAL k_E   = 1e-4;
	//	REAL lmd_E = 1e+4; // lmd_E = 1 / k_E

	/* 1. linear type */
		//return - q * Ey * y *(1 - LE_INV / 2 * y);
	/* 3. sinusoidal type E ~ + cos */
		//return - q * Ey / k_E * sin(k_E * y);
		/* 3+. 1/k_E -> lmd_Eとして小さな数が分母にくることを回避 */
		//return - q * Ey * lmd_E * sin(y / lmd_E);
	/* 3+.sinusoidal type E ~ + sin */
	//	return + q * Ey / k_E * cos(k_E * y);
	}
#endif

