#pragma once  // ヘッダファイルにはこれを書く

__device__ __host__ void FDM (int II, CMPLX *psi, REAL hx, REAL hy, int Nx2,
	CMPLX *dpsi_dx, CMPLX *dpsi_dy, CMPLX *d2psi_dx2, CMPLX *d2psi_dy2 )
{	
	REAL hx2 = Square( hx );
	REAL hy2 = Square( hy );
	#ifdef CUDACOMPLEX_H
	*dpsi_dx   = ( psi[II+1  ] - psi[II-1  ] ) / ( 2 * hx );
	*dpsi_dy   = ( psi[II+Nx2] - psi[II-Nx2] ) / ( 2 * hy );
	*d2psi_dx2 = ( psi[II+1  ] + psi[II-1  ] - 2 * psi[II] ) / hx2;
	*d2psi_dy2 = ( psi[II+Nx2] + psi[II-Nx2] - 2 * psi[II] ) / hy2;
	#else
	*dpsi_dx = hipCdiv( hipCsub( psi[II+1  ], psi[II-1  ] ), mkCMPLX( 2 * hx, 0 ) );
	*dpsi_dy = hipCdiv( hipCsub( psi[II+Nx2], psi[II-Nx2] ), mkCMPLX( 2 * hy, 0 ) );
	*d2psi_dx2 = hipCdiv( hipCsub( hipCadd( psi[II+1  ], psi[II-1  ] ), hipCmul( mkCMPLX( 2, 0 ), psi[II] ) ), mkCMPLX( hx2, 0 ) );
	*d2psi_dy2 = hipCdiv( hipCsub( hipCadd( psi[II+Nx2], psi[II-Nx2] ), hipCmul( mkCMPLX( 2, 0 ), psi[II] ) ), mkCMPLX( hy2, 0 ) );
	#endif
}

