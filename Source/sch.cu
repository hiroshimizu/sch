#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <cutil_inline.h>
//#include"cudacomplex.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLAS_V2
#ifdef BLAS_V2
	#include <hipblas.h>
#else
	#include <hipblas.h>
#endif


//#define FieldParticle
//#define REAL4

#ifndef REAL4
//	#define	eps				5e-21
//	#define	eps				5e-31
	#define	eps				5e-99
	#define	REAL 				double
	#define	REAL2 			double2
	#ifdef CUDACOMPLEX_H
		#define	CMPLX			doublecomplex
		#define	mkCMPLX		make_cudacomplex
	#else
		#define	CMPLX			hipDoubleComplex
		#define	mkCMPLX		make_hipDoubleComplex
	#endif
	#define	SIZEZ				16
	#define	SIZED				 8
#else
	#define	eps				5e-16
	#define	REAL 				float
	#define	REAL2 			float2
	#ifdef CUDACOMPLEX_H
		#define	CMPLX			singlecomplex
		#define	mkCMPLX		make_cudacomplex
	#else
		#define	CMPLX			hipDoubleComplex
		#define	mkCMPLX		make_hipComplex
	#endif
	#define	SIZEZ				8
	#define	SIZED				4
	#define	hipCadd			hipCaddf
	#define	hipCsub			hipCsubf
	#define	hipCmul			hipCmulf
	#define	hipCdiv			hipCdivf
	#define	hipCabs			hipCabsf
	#define	hipConj			hipConjf
	#define	hipCreal			hipCrealf
	#define	hipCimag			hipCimagf
	#define	hipblasDasum	hipblasSasum
	#define	hipblasDznrm2	hipblasScnrm2
#endif

#include "constant.cu"
#include "func.cu"
#include "FDM.cu"
#include "potential.cu"
#include "EXPECT.cu"
#include "write.cu"
#include "PHI_SOR.cu"
#include "timer.c"
#include "restart.c"

template < class T > T* host_allocate(size_t size)
{//return static_cast<T*>(malloc(sizeof(T)*size));
	T *ptr; hipHostAlloc( &ptr, sizeof(T) *size, hipHostMallocDefault );
	return ptr;
}

template < class T > void host_release(T* ptr) {	/*	free(ptr);*/	hipHostFree(ptr);}

//----------------------------------------------------------------------------
//------------------------------main------------------------------------------
//----------------------------------------------------------------------------
int main ( int argc, char** argv) // Program to solve the Schrodinger equation
//{	cutilDeviceInit(argc,argv);		//	hipSetDevice(1);
{	int devID = findCudaDevice(argc, (const char **)argv); printf("devID =%d\n",devID);
	std::string filename = (argc>=2) ? argv[1] : "sch.inp";

	StopWatchInterface *timer = 0;
	SetTimer(&timer);
	#ifdef BLAS_V2
		hipblasStatus_t stat;
		hipblasHandle_t handle;
		stat = hipblasCreate(&handle);
		if(stat != HIPBLAS_STATUS_SUCCESS)
		{	printf("CUBLAS initialization failed\n"); return EXIT_FAILURE; }
	#endif
	int N_rec		;
	int N_rec_rho	;
	int N_step		;
	REAL hx     	;
	REAL hy     	;
	REAL Lx      	;
	REAL Ly      	;
	REAL omg    	;
	REAL v0x    	;
	REAL v0y    	;
	REAL Bz			;
	REAL m			;
	REAL q			;
	REAL x0		   ;
	REAL y0		   ;
	REAL vSIZE	   ;
	REAL v0_	   ;
	REAL Bz_    	;
	REAL mp_	   ;
	REAL e_     	;
	REAL h_bar_ 	;

	if( !read_parameter(filename,
		N_rec, N_rec_rho, N_step, hx, hy, Lx, Ly,
		omg, v0x, v0y, Bz, m, q, x0, y0,
		vSIZE, v0_, Bz_, mp_, e_, h_bar_) )	return -1;

	REAL tau    = 5 * 2 * pi / N_step;
//	REAL tau    =     2 * pi / N_step;
	REAL hh     = hx * hy;
	REAL rl_    = mp_ * v0_ / (e_ * Bz_);
	REAL sgm    = sqrt( h_bar_ / (e_ * Bz_) ) * sqrt( 1 / ( q * Bz ) ) / rl_;
	REAL sgm2   = Square( sgm );
	REAL h_bar  = h_bar_ / (mp_ * v0_ * rl_);
	//REAL k0x    = m * v0x / h_bar;
	REAL k0x    =(m * v0x - q*Bz*y0) / h_bar;
	REAL k0y    = m * v0y / h_bar;
	int Nx 		= int( Lx / hx );
	int Ny 		= int( Ly / hy );
	int Nx2		= 2 * Nx + 1;
	int Ny2		= 2 * Ny + 1;
	int NG		= Nx2 * Ny2;
	//double N_t_s    =     N_step * m / (q * Bz) * 1050 / 100;
	//double N_t_s    =     N_step * m / (q * Bz) * 105 / 100 * (0.5 / 5);	//打ち切る用
	double N_t_s    =     N_step * m / (q * Bz) * 105 / 100 * (5 / 5);	// 5周用
	int    N_t_step = int(N_t_s);
	//N_t_step = 40;	// 打ちきり用
	int k_start = 1;

	CMPLX mvx_av, mvy_av, Px_av;
	CMPLX var_mvx, var_mvy, var_mv, var_Px, var_Py, var_P;
	REAL x_av, y_av, var_x, var_y, var_r;
	REAL qAx_av;
	REAL E0, EN;
	REAL err;
	FILE *interrupt;

	printf("magnetic length = %e\n", sgm);
	printf("Nx2 = %d Ny2 = %d hx = %e hy = %e NG = %d\n", Nx2, Ny2, hx, hy, NG);
	printf("h_bar = %e\n", h_bar);

	#ifndef FieldParticle
		int s3 = sizeof(CMPLX)+sizeof(CMPLX)+sizeof(REAL);	// sizeof Psi + Phi + Res
															// s3=20(REAL4),s3=40(double)
	#else
		int s3 = sizeof(CMPLX)+sizeof(CMPLX)+sizeof(REAL)+sizeof(REAL);	// sizeof Psi + Phi + Res + u
																		// s3=24(REAL4),s3=48(double)
	#endif
	int Nb = int( vSIZE / ( s3 * Nx2 ) - 2 );	// 一度に計算できるだろう行数
	printf("Nb = %d\n", Nb);
	int Nb2 = ( Ny2 - 2 ) / Nb;
	if ( Nb2 == 0 ) Nb = Ny2 - 2;	// Nb2 = 0 ならば一度に全部計算できる
	int Nb_ = Nb + 2;			// デバイスに一度に送る行数（基本）
	int Nb3_= Ny2 - Nb * Nb2;	// デバイスに最後に送る行数（余りの行）
	int Nb3 = Nb3_ - 2;			// 最後に計算する行数（余りの行）

	#ifdef BLAS_V2
		REAL var_tmp = 0;
	#endif

	printf("Ny2= %d Nb= %d Nb2= %d Nb3= %d Nb_= %d\n", Ny2, Nb, Nb2, Nb3, Nb_);
	// Ny2=全行数, Nb=一度に計算する行数

	//----------------------------------------------------------------------------
	REAL *x =(REAL*) host_allocate<REAL>(Nx2);
	REAL *y =(REAL*) host_allocate<REAL>(Ny2);
	for (int ix = 0; ix < Nx2; ix++){ x[ix] = hx * (ix-Nx); }
	for (int iy = 0; iy < Ny2; iy++){ y[iy] = hy * (iy-Ny); }

	#ifdef FieldParticle
		REAL *u = (REAL*) host_allocate< REAL> (NG); if( u == 0 ) { printf("%d Error\n", SIZED * NG); return 0; }
		REAL Eps0 = 8.8542e-12 / Square(e_) * mp_ * Square(v0_) * rl_;	// 真空の誘電率
		printf("Eps0= %e\n", Eps0);
		PotEne( u, x, y, q, sgm, Eps0, Nx2, Ny2 );
		pot( Nx2, Ny2, x, y, u );
	#endif

	CMPLX *Psi = (CMPLX*) host_allocate<CMPLX>(NG); if( Psi == 0 ) { printf("%d Error\n", SIZEZ * NG); return 0; }
	printf("host Psi\n");
	CMPLX *Phi = (CMPLX*) host_allocate<CMPLX>(NG); if( Phi == 0 ) { printf("%d Error\n", SIZEZ * NG); return 0; }
	printf("host Phi\n");

	//---------- LOAD Wavefunction ----------
	if( load_wavefunction(N_t_step, NG, &k_start, Psi, &E0) == 0 ) // 真/偽=1/0
	{	// *.binファイルが見つからない
		// Initialize the wavefunction *********************
		REAL Norm_Psi = 1 / ( sgm * sqrt(pi) );
		REAL Sum_Psi = 0;
		for(	int iy = 0; iy < Ny2; iy++) { REAL dy = y[iy] - y0; REAL dy2 = Square(dy);
			for(int ix = 0; ix < Nx2; ix++) { REAL dx = x[ix] - x0; REAL dx2 = Square(dx);
				int II = ix + Nx2 * iy;
				REAL aaa = Norm_Psi * exp( - ( dx2 + dy2 ) / ( 2 * sgm2 ) );
			//	REAL aaa = Norm_Psi * exp( - (       dy2 ) / ( 2 * sgm2 ) );
				REAL bbb = k0x * x[ix] + k0y * y[iy];
				#ifdef CUDACOMPLEX_H
				Psi[II].real() = aaa * cos( bbb );
				Psi[II].imag() = aaa * sin( bbb );
				Sum_Psi  += Square( Psi[II].real() ) + Square( Psi[II].imag() );
				#else
				Psi[II].x = aaa * cos( bbb );
				Psi[II].y = aaa * sin( bbb );
				Sum_Psi  += Square( hipCabs(Psi[II]) );
				#endif
			}
		}
		Sum_Psi *= hh;	printf("Rho     =%20.16f\n", Sum_Psi );
		// 自然境界条件	begin
		#ifdef CUDACOMPLEX_H
		printf("psi[0]={%11.3e,%11.3e} before BC\n", Psi[0].real(), Psi[0].imag());
		#else
		printf("psi[0]={%11.3e,%11.3e} before BC\n", hipCreal(Psi[0]), hipCimag(Psi[0]));
		#endif
		for( int iy = 0; iy < Ny2  ; iy++)
		{	int II, ix;
			#ifdef CUDACOMPLEX_H
			ix =     0; II = ix + Nx2 * iy; Psi[II] = Psi[II+1] + Psi[II+1] - Psi[II+2];
			ix = Nx2-1; II = ix + Nx2 * iy; Psi[II] = Psi[II-1] + Psi[II-1] - Psi[II-2];
			#else
			ix =     0; II = ix + Nx2 * iy; Psi[II] = hipCsub( hipCadd( Psi[II+1], Psi[II+1] ), Psi[II+2] );
			ix = Nx2-1; II = ix + Nx2 * iy; Psi[II] = hipCsub( hipCadd( Psi[II-1], Psi[II-1] ), Psi[II-2] );
			#endif
		}
		for( int ix = 1; ix < Nx2-1; ix++)
		{	int II, iy;
			#ifdef CUDACOMPLEX_H
			iy =     0; II = ix + Nx2 * iy; Psi[II] = Psi[II+Nx2] + Psi[II+Nx2] - Psi[II+2*Nx2];
			iy = Ny2-1; II = ix + Nx2 * iy; Psi[II] = Psi[II-Nx2] + Psi[II-Nx2] - Psi[II-2*Nx2];
			#else
			iy =     0; II = ix + Nx2 * iy; Psi[II] = hipCsub( hipCadd( Psi[II+Nx2], Psi[II+Nx2] ), Psi[II+2*Nx2] );
			iy = Ny2-1; II = ix + Nx2 * iy; Psi[II] = hipCsub( hipCadd( Psi[II-Nx2], Psi[II-Nx2] ), Psi[II-2*Nx2] );
			#endif
		}
		#ifdef CUDACOMPLEX_H
		printf("psi[0]={%11.3e,%11.3e}  after BC\n", Psi[0].real(), Psi[0].imag());
		#else
		printf("psi[0]={%11.3e,%11.3e}  after BC\n", hipCreal(Psi[0]), hipCimag(Psi[0]));
		#endif
		// 自然境界条件	end
		REAL Sum_Psi_ = 0;
		for(	int iy = 0; iy < Ny2; iy++)
		{	for(int ix = 0; ix < Nx2; ix++)
			{	int II = ix + Nx2 * iy;
				#ifdef CUDACOMPLEX_H
				Psi[II]   = Psi[II] / sqrt(Sum_Psi);
				Sum_Psi_ += Square( Psi[II].real() ) + Square( Psi[II].imag() );
				#else
				Psi[II].x /= sqrt(Sum_Psi);
				Psi[II].y /= sqrt(Sum_Psi);
				Sum_Psi_  += Square( hipCabs(Psi[II]) );
				#endif
			}
		}
		Sum_Psi_ *= hh;	printf("Rho_bar =%20.16f\n", Sum_Psi_);
	/*
		// cublasの使い方例
		{	REAL *s = (REAL*) malloc( SIZED * NG );
			for(	int iy = 0; iy < Ny2; iy++)
			{	for(int ix = 0; ix < Nx2; ix++)
				{	int II = ix + Nx2 * iy;
					s[II] = Square( hipCabs(Psi[II]) ) * hh;
				}
			}
			hipblasStatus_t stat;
			REAL *S;
			cublasInit(); // printf("sizeof(*s) = %d\n", sizeof(*s));
			stat = cublasAlloc ( NG, SIZED, (void**) &S );	// printf ("%s\n", stat);
			if ( stat != HIPBLAS_STATUS_SUCCESS) { printf ("device memory allocation failed"); return 1; }
			hipblasSetVector( NG, SIZED, s, 1, S, 1);	//printf ("%s\n", cublasGetError());
			printf("blasasum=%20.16f, blas error = %s\n", hipblasDasum( NG, S, 1), cublasGetError());
			cublasFree(S);
			cublasShutdown();
			free(s);
		}
	*/
		//-------------init judge---------------------------------------------------
		EXPECT( Psi, x, y,
		#ifdef FieldParticle
		                   u,
		#endif
				&mvx_av, &mvy_av, &Px_av, &E0, &var_mv, &var_mvx, &var_mvy, &var_P, &var_Px, &var_Py,
				&x_av, &y_av, &var_r, &var_x, &var_y, &err, &qAx_av,
				Bz, m, q, Nx2, Ny2, hx, hy, h_bar);

		#ifdef CUDACOMPLEX_H
		printf("<mvx> = %24.16e\n", mvx_av.real());
		printf("<qAx> = %24.16e\n", qAx_av       );
		printf("<P_x> = %24.16e\n",  Px_av.real());
		#else
		printf("<mvx> = %24.16e\n", mvx_av.x     );
		printf("<qAx> = %24.16e\n", qAx_av       );
		printf("<P_x> = %24.16e\n",  Px_av.x     );
		#endif

		EN = E0;
		Record( 0, tau, E0, EN, var_mvx, var_mvy, var_mv, mvx_av, mvy_av, var_Px, var_Py, var_P, x_av, y_av, var_x, var_y, var_r, Px_av, err);
		rho(0, Nx2, Ny2, hh, x, y, Psi);

	//	OUTに一初めの~~~.rhoを書いたところで止めるためのreturn 0;
	//	return 0;
	//
		printf("      k itr     <x>     <y>  var[r]     <u>     <v>  var[p]           <Px>          EN       EN-E0       error\n");
		printf("%7d %3d %7.3f %7.3f %7.3f %7.3f %7.3f %7.3f %14.6e %11.3e %11.3e %11.3e\n",
		#ifdef CUDACOMPLEX_H
				 0, 0, x_av, y_av, var_r, mvx_av.real(), mvy_av.real(), var_mv.real(), Px_av.real(), E0, EN-E0, err);
		#else
				 0, 0, x_av, y_av, var_r, mvx_av.x     , mvy_av.x     , var_mv.x     , Px_av.x     , E0, EN-E0, err);
		#endif
		/********** SAVE Initial Wavefunction **********/
		printf("Save Initial Wavefunction, k = %d\n", 0);
		save_wavefunction(NG, 0, Psi, E0);
		/***********************************************/
	}

	#ifdef FieldParticle
	REAL     *u_device; hipMalloc( (void**) &u_device,   SIZED * Nx2 * Nb_ );
	#endif
	CMPLX  *Psi_device; hipMalloc( (void**) &Psi_device, SIZEZ * Nx2 * Nb_ );
	CMPLX  *Phi_device; hipMalloc( (void**) &Phi_device, SIZEZ * Nx2 * Nb_ );
	REAL   *Res_device; hipMalloc( (void**) &Res_device, SIZED * Nx2 * Nb_ );

//	int THREADS =  224;
//	int THREADS =  256;
	int THREADS =  512;
//	int THREADS = 1024;	// for GT-480
	int GRIDS = Nx2 * Nb_ / THREADS; if ( Nx2 * Nb_ % THREADS != 0 ) GRIDS++;
	dim3 grid( GRIDS );
	dim3 threads(THREADS);

	//*************** Set coeffcient ***********************
	#ifdef CUDACOMPLEX_H
	CMPLX beta  =        { 0, tau * h_bar / ( 4 * m ) };
	CMPLX gamma =        {    tau / ( 2 * m ),    0   };
	CMPLX zeta  =        { 0, tau / ( 4 * m * h_bar ) };
	CMPLX xi    =        { 0, tau / ( 2 *     h_bar ) };
	#else
	CMPLX beta  = mkCMPLX( 0, tau * h_bar / ( 4 * m ) );
	CMPLX gamma = mkCMPLX(    tau / ( 2 * m ),    0   );
	CMPLX zeta  = mkCMPLX( 0, tau / ( 4 * m * h_bar ) );
	CMPLX xi    = mkCMPLX( 0, tau / ( 2 *     h_bar ) );
	#endif
	//*************** end Set coeffcient *******************
	SetConst( Nb, Nx2, Ny2, Nb3_, Nx, Ny, sgm2, pi, hx, hy, hh, omg, x0, y0, Bz, q, beta, gamma, zeta, xi);
	//
	if( Nb2 == 0 ) {	// すべてビデオメモリに収まる
		hipMemcpy( Psi_device, Psi, SIZEZ * NG, hipMemcpyHostToDevice);
		#ifdef FieldParticle
		hipMemcpy(   u_device,   u, SIZED * NG, hipMemcpyHostToDevice);
		#endif
	}
	//
//	cublasInit();
//	******* Total computing loop *************************
	printf("k: %d -> %d\n", k_start, N_t_step);
	int k;
	for(k = k_start; k <= N_t_step; k++)
	{		for(	int k2 = 1; k2 <= Nb2; k2++)	// すべてビデオメモリに収まらない
		{	int N_field = k2 - 1;
			hipMemcpy ( Psi_device, Psi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb_ , hipMemcpyHostToDevice);
			#ifdef FieldParticle
			hipMemcpy (   u_device,   u + Nx2 * Nb * N_field, SIZED * Nx2 * Nb_ , hipMemcpyHostToDevice);
			PHI <<< grid, threads >>> ( Psi_device, Phi_device, u_device, N_field );
			#else
			PHI <<< grid, threads >>> ( Psi_device, Phi_device,           N_field );
			#endif
			hipMemcpy ( Phi+ Nx2 * Nb * N_field + Nx2, Phi_device + Nx2, SIZEZ * Nx2 * Nb, hipMemcpyDeviceToHost);
		}
		if( Nb3 > 0 )
		{	int N_field = Nb2;
			if ( Nb2 != 0 ) {
				hipMemcpy ( Psi_device, Psi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb3_, hipMemcpyHostToDevice);
				#ifdef FieldParticle
				hipMemcpy (   u_device,   u + Nx2 * Nb * N_field, SIZED * Nx2 * Nb3_, hipMemcpyHostToDevice);
				#endif
			}
			#ifdef FieldParticle
				PHI <<< grid, threads >>> ( Psi_device, Phi_device, u_device, N_field );
			#else
				PHI <<< grid, threads >>> ( Psi_device, Phi_device,           N_field );
			#endif
hipDeviceSynchronize();
			if ( Nb2 != 0 )
				hipMemcpy ( Phi + Nx2 * Nb * N_field + Nx2, Phi_device + Nx2, SIZEZ * Nx2 * Nb3, hipMemcpyDeviceToHost);
		}

		//---------start while--------------------------------------------------------
		REAL var = 2 * eps, var0 = 1e99; int itr = 0;
		while( var > eps )
		{	itr++;	var = 0;
			for(int k2 = 1; k2 <= Nb2; k2++)
			{	// 領域の分割がある場合
				int N_field = k2 - 1;
				hipMemcpy ( Phi_device, Phi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb_, hipMemcpyHostToDevice);
				hipMemcpy ( Psi_device, Psi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb_, hipMemcpyHostToDevice);
				#ifdef FieldParticle
				hipMemcpy ( u_device, u + Nx2 * Nb * N_field, SIZED * Nx2 * Nb_, hipMemcpyHostToDevice);
				#endif
				if (1)	/* 1 <- 0, on May 18, 2015: oikawa */
				{// 1
					#ifdef FieldParticle
						SOR <<< grid, threads >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
					#else
						SOR <<< grid, threads >>> ( Psi_device, Phi_device, Res_device, N_field );
					#endif
hipDeviceSynchronize();
				}
				else
				{// 0		/*Does not work on host-b (CUDA 7.0) on May 18, 2015: oikawa */
					int BLOCKS = ( Ny2 % THREADS == 0  ) ? Ny2 / THREADS : Ny2 / THREADS  + 1;
					#ifdef FieldParticle
						sor0<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
hipDeviceSynchronize();
						sor1<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
					#else
						sor0<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device,           N_field );
hipDeviceSynchronize();
						sor1<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device,           N_field );
					#endif
hipDeviceSynchronize();
				}
				hipMemcpy ( Psi + Nx2 * Nb * N_field + Nx2, Psi_device + Nx2, SIZEZ * Nx2 * Nb , hipMemcpyDeviceToHost);
				#ifdef BLAS_V2
					stat = hipblasDasum(handle, Nx2 * Nb, Res_device + Nx2, 1, &var_tmp);
					var += var_tmp;
				#else
					var += hipblasDasum ( Nx2 * Nb , Res_device + Nx2, 1) ;
				#endif
			}
			if( Nb3 > 0 )
			{	int N_field = Nb2;
				if ( Nb2 != 0 )
				{	( hipMemcpy ( Phi_device, Phi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb3_, hipMemcpyHostToDevice));
					( hipMemcpy ( Psi_device, Psi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb3_, hipMemcpyHostToDevice));
					#ifdef FieldParticle
						( hipMemcpy ( u_device, u + Nx2 * Nb * N_field, SIZED * Nx2 * Nb3_, hipMemcpyHostToDevice));
					#endif
				}
				if (1)
				{// 1
					#ifdef FieldParticle
						SOR <<< grid, threads >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
					#else
						SOR <<< grid, threads >>> ( Psi_device, Phi_device, Res_device, N_field );
					#endif
hipDeviceSynchronize();
				}
				else
				{// 0
					int BLOCKS = ( Ny2 % THREADS == 0  ) ? Ny2 / THREADS : Ny2 / THREADS  + 1;
					#ifdef FieldParticle
						sor0<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
hipDeviceSynchronize();
						sor1<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device, u_device, N_field );
					#else
						sor0<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device,           N_field );
hipDeviceSynchronize();
						sor1<<< BLOCKS, THREADS >>> ( Psi_device, Phi_device, Res_device,           N_field );
hipDeviceSynchronize();
					#endif
				}
				if ( Nb2 != 0 )
					hipMemcpy ( Psi + Nx2 * Nb * N_field + Nx2, Psi_device + Nx2, SIZEZ * Nx2 * Nb3, hipMemcpyDeviceToHost );
				// begin BLAS
				#ifdef BLAS_V2
					stat = hipblasDasum (handle, Nx2 * Nb3 , Res_device + Nx2, 1, &var_tmp) ;
					var += var_tmp;
				#else
					var += hipblasDasum ( Nx2 * Nb3 , Res_device + Nx2, 1) ;
				#endif
				//	end BLAS
				hipDeviceSynchronize();
			}
			var /= NG;
			int NotNumber = isnan(var);
			if ( itr == 99 || NotNumber != 0 )
			{	printf("+++++ Abnormal end +++++\nOVER itr = %d, NAN = %d\n", itr, NotNumber);
				/***** 終了処理 **************************************************************/
				fclose(interrupt);	//	中断チェック用のFILE*変数をクローズする
				hipFree(Psi_device); hipFree(Phi_device); hipFree(Res_device);
				#ifdef FieldParticle
				hipFree(u_device);
				#endif
				hipDeviceReset();
				// -----------------------------------------------------------------------------
				host_release(x); host_release(y); host_release(Psi); host_release(Phi);
				#ifdef FieldParticle
				host_release(u);
				#endif
				/*****************************************************************************/
				exit(1);
			}
			hipDeviceSynchronize();
			//	***** for test	*******************************
			//	if( NotNumber == 0 ) printf("var = %e\n", var);
			//	******************************************************
			//printf("k=%3d, itr = %2d, var = %12.3e\n", k, itr, var);
			if( var > var0 / 2 )	break;
			var0 = var;
		}
		//----- end while ---------------------------------------------------------------------

		if (  Nb2     == 0 ) ( hipMemcpy ( Psi, Psi_device, SIZEZ * NG, hipMemcpyDeviceToHost) );
		// 毎回必ず、デバイスからホストへPsiを戻す
		/********** 期待値らを出力する **********/
		if( k % N_rec == 0 ) {
			EXPECT( Psi, x, y,
			#ifdef FieldParticle
			                   u,
			#endif
					&mvx_av, &mvy_av, &Px_av, &EN, &var_mv, &var_mvx, &var_mvy, &var_P, &var_Px, &var_Py,
					&x_av, &y_av, &var_r, &var_x, &var_y, &err, &qAx_av,
					Bz, m, q, Nx2, Ny2, hx, hy, h_bar);
			Record( k, tau, E0, EN, var_mvx, var_mvy, var_mv, mvx_av, mvy_av, var_Px, var_Py, var_P, x_av, y_av, var_x, var_y, var_r, Px_av, err);
			printf("%7d %3d %7.3f %7.3f %7.3f %7.3f %7.3f %7.3f %14.6e %11.3e %11.3e %11.3e\n",
			#ifdef CUDACOMPLEX_H
					 k, itr, x_av, y_av, var_r, mvx_av.real(), mvy_av.real(), var_mv.real(), Px_av.real(), EN, EN-E0, err);
			#else
					 k, itr, x_av, y_av, var_r, mvx_av.x, mvy_av.x, var_mv.x, Px_av.x, EN, EN-E0, err);
			#endif
			if ( k == N_t_step )
				printf("      k itr     <x>     <y>  var[r]     <u>     <v>  var[p]           <Px>          EN       EN-E0       error\n");

			// ここにあったらN_rec_rhoはN_recの倍数でないと一切出力されないのでは?
		}	//printf("k=%7d, itr = %2d, var0 = %12.3e, var = %12.3e\n", k, itr, var0, var);
		/********** 粒子保存の確認 **********/
		if( k % 20 == 0 )		// 確認が終わったら\nを\rにする
		{	if( Nb2 == 0 )
			{	//	領域分割なし
				#ifdef BLAS_V2
					REAL Dznrm2_tmp;
					stat = hipblasDznrm2(handle, NG,
					#ifdef CUDACOMPLEX_H
						(hipDoubleComplex*)
					#endif
					Psi_device, 1, &Dznrm2_tmp);
					printf("k=%7d, itr=%3d, residue=%11.3e, error using blasnrm2=%11.3e\n",
						     k, itr, var, Square(Dznrm2_tmp) * hh - 1 );
				#else
//					printf("blasasum=%11.3e\n", Square(hipblasDasum(2*NG,(REAL*)Psi_device, 1)) * hh - 1 );
					printf("k=%7d, itr=%3d, residue=%11.3e, error using blasnrm2=%11.3e\n",
						     k, itr, var, Square(hipblasDznrm2(NG,
					#ifdef CUDACOMPLEX_H
						 		(hipDoubleComplex*)
					#endif
					    		Psi_device, 1))* hh - 1 );
				#endif
				hipDeviceSynchronize();
				/*
				printf("k=%7d, itr=%3d, residue=%11.3e, error using blasdot =%25.16e\n",
					    k, itr, var, hipblasDdot ( 2*NG, (REAL*)Psi_device, 1, (REAL*)Psi_device, 1)*hh-1 );
				hipDeviceSynchronize();
				*/
			}
			else
			{	// 領域分割あり
				// ************ cublas使って粒子保存を見る場合はこっち。
				REAL blasnrm2 = 0;	//REAL blasdot  = 0;
				#ifdef BLAS_V2
					REAL blasDznrm2_tmp1 = 0;
					stat = hipblasDznrm2( handle, Nx2 * (Nb3 + 1),
					#ifdef CUDACOMPLEX_H
						(hipDoubleComplex*)
					#endif
					(Psi_device + Nx2), 1, &blasDznrm2_tmp1);
					blasnrm2 = blasDznrm2_tmp1 * blasDznrm2_tmp1;
				#else
					blasnrm2 = Square(hipblasDznrm2( Nx2 * (Nb3 + 1),
					#ifdef CUDACOMPLEX_H
						(hipDoubleComplex*)
					#endif
					(Psi_device + Nx2), 1) ); // 最後のブロック＋最後の行
					/*
					#ifdef CUDACOMPLEX_H
					blasnrm2 = Square(hipblasDznrm2( Nx2 * (Nb3 + 1), (hipDoubleComplex*)(Psi_device + Nx2), 1) ); // 最後のブロック＋最後の行
					#else
					blasnrm2 = Square(hipblasDznrm2( Nx2 * (Nb3 + 1),                    Psi_device + Nx2 , 1) ); // 最後のブロック＋最後の行
					#endif
					*/
				#endif
				/*
				hipDeviceSynchronize();
				blasdot  =      hipblasDdot( 2 * Nx2 * (Nb3 + 1), (REAL*)(Psi_device + Nx2), 1, (REAL*)(Psi_device + Nx2), 1);
				hipDeviceSynchronize();
				*/
				hipMemcpy ( Psi_device, Psi, SIZEZ * Nx2 * Nb_, hipMemcpyHostToDevice);
				#ifdef BLAS_V2
					REAL blasDznrm2_tmp2 = 0;
					stat = hipblasDznrm2( handle, Nx2 * (Nb + 1),
					#ifdef CUDACOMPLEX_H
						(hipDoubleComplex*)
					#endif
					(Psi_device + Nx2), 1, &blasDznrm2_tmp2);
					blasnrm2 += blasDznrm2_tmp2 * blasDznrm2_tmp2;
					#else
					blasnrm2 += Square(hipblasDznrm2( Nx2 * (Nb + 1),
					#ifdef CUDACOMPLEX_H
						(hipDoubleComplex*)
					#endif
					Psi_device, 1));		 // 最初のブロック＋最初の行
				/*
				#ifdef CUDACOMPLEX_H
				blasnrm2 += Square(hipblasDznrm2( Nx2 * (Nb + 1), (hipDoubleComplex*)Psi_device, 1));		 // 最初のブロック＋最初の行
				#else
				blasnrm2 += Square(hipblasDznrm2( Nx2 * (Nb + 1),                   Psi_device, 1));		 // 最初のブロック＋最初の行
				#endif
				*/
				#endif
				hipDeviceSynchronize();
				/*
				blasdot  +=        hipblasDdot( 2 * Nx2 * (Nb + 1), (REAL*)Psi_device, 1, (REAL*)Psi_device, 1);
				hipDeviceSynchronize();
				*/
				for(int k2 = 2; k2 <= Nb2; k2++)
				{	int N_field = k2 - 1;
					hipMemcpy ( Psi_device, Psi + Nx2 * Nb * N_field, SIZEZ * Nx2 * Nb_, hipMemcpyHostToDevice);
					#ifdef BLAS_V2
						REAL blasDznrm2_tmp3 = 0;
						stat = hipblasDznrm2( handle,  Nx2 * Nb,
						#ifdef CUDACOMPLEX_H
							(hipDoubleComplex*)
						#endif
						(Psi_device + Nx2), 1, &blasDznrm2_tmp3);	// 中間のブロック
						blasnrm2 += blasDznrm2_tmp3 * blasDznrm2_tmp3;
					#else
						blasnrm2 += Square(hipblasDznrm2( Nx2 * Nb,
						#ifdef CUDACOMPLEX_H
							(hipDoubleComplex*)
						#endif
						(Psi_device + Nx2), 1));	// 中間のブロック
						/*
						#ifdef CUDACOMPLEX_H
						blasnrm2 += Square(hipblasDznrm2( Nx2 * Nb, (hipDoubleComplex*)(Psi_device + Nx2), 1));	// 中間のブロック
						#else
						blasnrm2 += Square(hipblasDznrm2( Nx2 * Nb, Psi_device + Nx2, 1));	// 中間のブロック
						#endif
						*/
					#endif
					hipDeviceSynchronize();
					/*
					blasdot  +=      hipblasDdot( 2 * Nx2 * Nb, (REAL*)(Psi_device + Nx2), 1, (REAL*)(Psi_device + Nx2), 1);
					hipDeviceSynchronize();
					*/
				}
				printf("k=%7d, itr=%3d, blasnrm2=%11.3e\n", k, itr, blasnrm2 * hh - 1);
				//printf("k=%7d, itr=%3d, blasdot =%25.16e\n", k, itr, blasdot  * hh - 1);

//				************ cublas使いたくなくて、どうしても粒子保存見たければ使う。
/*				REAL asum = 0;
				for(		int iy = 0; iy < Ny2; iy++)
				{	for(	int ix = 0; ix < Nx2; ix++)
					{	int II = ix + Nx2 * iy;
						#ifdef CUDACOMPLEX_H
						asum += Square( Psi[II].real() ) + Square( Psi[II].imag() );
						#else
						asum += Square( hipCabs( Psi[II] ) );
						#endif
					}
				}
				printf("k=%7d, itr=%3d, asum=%25.16e\r", k, itr, asum * hh - 1);
*/
			}
		}
		/********** ディレクトリOUTに精度を落とした確率密度関数を出力する **********/
		if ( k % N_rec_rho == 0 )
		{	rho ( k, Nx2, Ny2, hh, x, y, Psi);
			/********** Interruption **********/
			if( (interrupt = fopen("SAVE/interruption", "r")) != NULL )
			{	printf(" Interruption, k = %d\n", k); // このタイミングなら今回の波動関数
				fclose(interrupt);	//	中断チェック用のFILE*変数をクローズする
				break;
			}
			/**********************************/
		}
		/********** 10500ごとに定期的に波動関数を出力させる **********/
		// 典型的なパラメータで105000は5周程度、10500は半周くらい
		if(k % 10500 == 0)
		{	printf("Save Wavefunction, k = %d\n", k);
			save_wavefunction(NG, k, Psi, E0);
		}
		/********************************************************/
		fflush(stdout);
	}	printf("Elapsed TIME = %f sec.\n", EndTimer(&timer) / 1000 );
	/*****************************************************************************/
	/***** 終了処理 **************************************************************/
	/*****************************************************************************/
	//---------- SAVE last wavefunction --------------
	//printf("*** Save Last wavefunction ***\n");
	//	普通にループを終了したならばk==N_t_step+1となっているので補正する
	if( k == N_t_step+1 ) k--;
	save_wavefunction(NG, k, Psi, E0);
	//---------- END SAVE last wavefunction ----------
	#ifdef BLAS_V2
		fprintf(stdout, "hipblasDestroy(blas4.0)\n"); hipblasDestroy(handle);
	#endif
	/* free allocation (GPU) */
	hipFree(Psi_device); hipFree(Phi_device); hipFree(Res_device);
	#ifdef FieldParticle
		hipFree(u_device);
	#endif
	hipDeviceReset();
	// -----------------------------------------------------------------------------
	/* free allocation (CPU) */
	host_release(x); host_release(y); host_release(Psi); host_release(Phi);
	#ifdef FieldParticle
		host_release(u);
	#endif
	return 0;
}
